#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>


#define N 100000000
#define MAX_ERR 1e-6

// Uncomment ONE of the following lines to select the kernel to run:
// #define KERNEL_SINGLE
// #define KERNEL_MULTITHREAD
#define KERNEL_MULTIBLOCK

// ------------------- Kernels -------------------

// 1️⃣ Single-thread kernel
__global__ void vector_add_kernel(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

// 2️⃣ Single-block multi-thread kernel
__global__ void vector_add_kernel_multithreads(float *out, float *a, float *b, int n) {
    int idx = threadIdx.x;
    int stride = blockDim.x;
    for(int i = idx; i < n; i += stride) {
        out[i] = a[i] + b[i];
    }
}

// 3️⃣ Multi-block multi-thread kernel
__global__ void vector_add_kernel_multiblocks(float *out, float *a, float *b, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = idx; i < n; i += stride) {
        out[i] = a[i] + b[i];
    }
}

// ------------------- Utility Functions -------------------
void verify(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        if(fabs(out[i] - (a[i] + b[i])) > MAX_ERR){
            printf("Mismatch at %d: %f != %f\n", i, out[i], a[i]+b[i]);
            exit(1);
        }
    }
}

// Print GPU memory usage
void printGPUMemUsage(const char *msg) {
    size_t free_bytes, total_bytes;
    hipMemGetInfo(&free_bytes, &total_bytes);
    printf("%s - GPU memory used: %.2f MB / %.2f MB\n", 
        msg, (total_bytes - free_bytes)/1024.0/1024.0, total_bytes/1024.0/1024.0);
}

// ------------------- Main -------------------
int main() {
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    // Allocate host memory
    a = (float*)malloc(sizeof(float)*N);
    b = (float*)malloc(sizeof(float)*N);
    out = (float*)malloc(sizeof(float)*N);

    // Initialize input vectors on host
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    // Print initial GPU memory usage
    printGPUMemUsage("Before device allocation");

    // Allocate device memory
    hipMalloc(&d_a, sizeof(float)*N);
    hipMalloc(&d_b, sizeof(float)*N);
    hipMalloc(&d_out, sizeof(float)*N);

    // Print GPU memory usage after allocation
    printGPUMemUsage("After device allocation");
    
    // Copy input vectors from host to device
    hipMemcpy(d_a, a, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float)*N, hipMemcpyHostToDevice);

    // CUDA event timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float ms;

#ifdef KERNEL_SINGLE
    cudaEventRecord(start);
    vector_add_kernel<<<1,1>>>(d_out, d_a, d_b, N);
    cudaEventRecord(stop);
#elif defined(KERNEL_MULTITHREAD)
    int blockSize = 256;
    cudaEventRecord(start);
    vector_add_kernel_multithreads<<<1, blockSize>>>(d_out, d_a, d_b, N);
    cudaEventRecord(stop);
#elif defined(KERNEL_MULTIBLOCK)
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1)/blockSize;
    hipEventRecord(start);
    vector_add_kernel_multiblocks<<<numBlocks, blockSize>>>(d_out, d_a, d_b, N);
    hipEventRecord(stop);
#else
    #error "Please define one kernel macro: KERNEL_SINGLE, KERNEL_MULTITHREAD, KERNEL_MULTIBLOCK"
#endif

    // Copy result back to host and verify
    hipEventSynchronize(stop);
    hipMemcpy(out, d_out, sizeof(float)*N, hipMemcpyDeviceToHost);
    verify(out, a, b, N);

#ifdef KERNEL_SINGLE
    printf("Single-thread kernel PASSED\n");
#elif defined(KERNEL_MULTITHREAD)
    printf("Single-block multi-thread kernel PASSED\n");
#elif defined(KERNEL_MULTIBLOCK)
    printf("Multi-block multi-thread kernel PASSED\n");
#endif

    hipEventElapsedTime(&ms, start, stop);
    printf("Kernel execution time: %f ms\n", ms);

    hipFree(d_a); hipFree(d_b); hipFree(d_out);
    free(a); free(b); free(out);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
